
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SoftClassNLLCriterion.cu"
#else

void THNN_(SoftClassNLLCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *total_weight) {
  THCUNN_check_dim_size(state, output, 1, 0, 1);
  THCUNN_check_dim_size(state, total_weight, 1, 0, 1);


  int n_dims = THCTensor_(nDimension)(state, input);
  int n_classes = THCTensor_(size)(state, input, n_dims - 1);
  int n_weights = 10; //THCIndexTensor_(size)(state, target, n_dims -1);

  if (weights) {
    THCUNN_assertSameGPU(
      state, 5, input, target, weights, output, total_weight
    );
  } else {
    THCUNN_assertSameGPU(
      state, 4, input, target, output, total_weight
    );
  }

  THArgCheck(n_dims <= 2 && n_dims > 0, 2, "vector or matrix expected");

  long batch_size = n_dims == 1 ? 1 : THCTensor_(size)(state, input, 0);
  long num_targets = THCudaLongTensor_size(state, target, 0);
  THArgCheck(batch_size == num_targets,
      2, "mismatch between the batch size of input (%ld) and that of target (%ld)",
      batch_size, num_targets);

  if (weights && THCTensor_(size)(state, weights, n_dims -1) != n_weights) {
    THCDescBuff s1 = THCTensor_(sizeDesc)(state, weights);
    THError("weight tensor should be defined for all %d targets "
            " but got weight tensor of shape: %s", n_weights, s1.str);
  }

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  real *input_data = THCTensor_(data)(state, input);
  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  THCIndex_t  *target_data = THCIndexTensor_(data)(state, target);
  real *output_data = THCTensor_(data)(state, output);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  if (THCTensor_(nDimension)(state, input) == 1) {
    cunn_SoftClassNLLCriterion_updateOutput_kernel1<real>
      <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
        output_data,
        total_weight_data,
        input_data,
        target_data,
        weights_data,
        sizeAverage,
        n_classes
    );

  } else if (THCTensor_(nDimension)(state, input) == 2) {
    cunn_SoftClassNLLCriterion_updateOutput_kernel<real, accreal>
      <<<1, NTHREADS, 0, THCState_getCurrentStream(state)>>>(
        output_data,
        total_weight_data,
        input_data,
        target_data,
        weights_data,
        sizeAverage,
        THCTensor_(size)(state, input, 0),
        THCTensor_(size)(state, input, 1),
        n_classes,
        n_weights
    );
  }
  THCudaCheck(cudaGetLastError());

  if (weights) {
    THCTensor_(free)(state, weights);
  }
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

void THNN_(SoftClassNLLCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradInput,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *total_weight) {

  int n_dims = THCTensor_(nDimension)(state, input);
  int n_classes = THCTensor_(size)(state, input, n_dims - 1);
  int n_weights = 10; //THCIndexTensor_(size)(state, target, n_dims -1);

  THArgCheck(THCTensor_(isContiguous)(state, gradInput), 4, "gradInput must be contiguous");

  if (weights) {
    THCUNN_assertSameGPU(
      state, 5, weights, input, target, gradInput, total_weight
    );
  }
  else {
    THCUNN_assertSameGPU(
      state, 4, input, target, gradInput, total_weight
    );
  }

  THArgCheck(n_dims <= 2 && n_dims > 0, 2, "vector or matrix expected");

  long batch_size = n_dims == 1 ? 1 : THCTensor_(size)(state, input, 0);
  long num_targets = THCudaLongTensor_size(state, target, 0);
  THArgCheck(batch_size == num_targets,
      2, "mismatch between the batch size of input (%ld) and that of target (%ld)",
      batch_size, num_targets);

  if (weights && THCTensor_(size)(state, weights, n_dims -1) != n_weights) {
    THCDescBuff s1 = THCTensor_(sizeDesc)(state, weights);
    THError("weight tensor should be defined for all %d targets "
            " but got weight tensor of shape: %s", n_weights, s1.str);
  }

  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  real *gradInput_data = THCTensor_(data)(state, gradInput);
  THCIndex_t  *target_data = THCIndexTensor_(data)(state, target);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  if (THCTensor_(nDimension)(state, input) == 1) {
    cunn_SoftClassNLLCriterion_updateGradInput_kernel1<real>
      <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
        gradInput_data,
        weights_data,
        target_data,
        total_weight_data,
        sizeAverage,
        n_classes
    );
  } else {
    cunn_SoftClassNLLCriterion_updateGradInput_kernel<real>
      <<<1, NTHREADS, 0, THCState_getCurrentStream(state)>>>(
        gradInput_data,
        target_data,
        weights_data,
        total_weight_data,
        sizeAverage,
        THCTensor_(size)(state, input, 0),
        THCTensor_(size)(state, input, 1),
        n_classes,
        n_weights
    );
  }
  THCudaCheck(cudaGetLastError());

  if (weights) {
    THCTensor_(free)(state, weights);
  }
  THCIndexTensor_(free)(state, target);
}

#endif
